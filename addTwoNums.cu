

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c){
  *c = *a + *b;
}

int main(){
  int n1=2, n2=3, n3;
  int *d1, *d2, *d3;
  int size = sizeof(int);
  hipMalloc((void **)&d1, size);
  hipMalloc((void **)&d2, size);
  hipMalloc((void **)&d3, size);
  hipMemcpy(d1, &n1, size, hipMemcpyHostToDevice);
  hipMemcpy(d2, &n2, size, hipMemcpyHostToDevice);
  add<<<1,1>>>(d1, d2, d3);
  hipMemcpy(&n3, d3, size, hipMemcpyDeviceToHost);
  printf("Sum of %d and %d = %d\n", n1, n2, n3);
  hipFree(d1); hipFree(d2); hipFree(d3);
  return 0;
}
