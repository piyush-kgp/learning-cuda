

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10


// F1
// __global__ void add(int *a, int *b, int *c){
//   int idx = blockIdx.x;
//   if (idx<N){
//     c[idx] = a[idx]+b[idx];
//   }
// }


//F2
// __global__ void add(int *a, int *b, int *c){
//   int idx = threadIdx.x;
//   if (idx<N){
//     c[idx] = a[idx]+b[idx];
//   }
// }


// F3
__global__ void add(int *a, int *b, int *c){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N){
    c[idx] = a[idx]+b[idx];
  }
}


int main(){
  int a[N], b[N], c[N];
  for (int i=0; i<N; i++){
    a[i] = rand()%100;
    b[i] = rand()%100;
  }
  int *d1, *d2, *d3;
  int size = sizeof(int);
  hipMalloc((void **)&d1, N*size);
  hipMalloc((void **)&d2, N*size);
  hipMalloc((void **)&d3, N*size);
  hipMemcpy(d1, a, N*size, hipMemcpyHostToDevice);
  hipMemcpy(d2, b, N*size, hipMemcpyHostToDevice);

  // add<<<N,1>>>(d1, d2, d3); //use with F1
  // add<<<1,N>>>(d1, d2, d3); //use with F2
  int nBlocks=2, nThreadsPerBlock=N/2;
  add<<<nBlocks,nThreadsPerBlock>>>(d1, d2, d3); //use with  F3

  hipMemcpy(c, d3, N*size, hipMemcpyDeviceToHost);
  hipFree(d1); hipFree(d2); hipFree(d3);
  for (int i=0; i<N; i++){
    printf("Sum of %d and %d = %d\n", a[i], b[i], c[i]);
  }
  return 0;
}
