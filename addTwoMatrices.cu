

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define ROWS 2
#define COLS 3


// F1
// __global__ void add(int *a, int *b, int *c){
//   int idx = blockIdx.x*blockDim.x + threadIdx.x;
//   if (idx<ROWS*COLS){
//     c[idx] = a[idx] + b[idx];
//   }
// }

// F2
__global__ void add(int *a, int *b, int *c){
  int col = blockIdx.x*blockDim.x;
  int row = blockIdx.y*blockDim.y;
  if (row<ROWS && col<COLS){
    int idx = row*COLS + col;
    c[idx] = a[idx] + b[idx];
  }
}

int main(){
  int a[ROWS][COLS], b[ROWS][COLS], c[ROWS][COLS];
  for (int i=0; i<ROWS; i++){
    for (int j=0; j<COLS; j++){
      a[i][j] = rand()%100;
      b[i][j] = rand()%100;
    }
  }
  int *d1, *d2, *d3;
  int size = sizeof(int);
  hipMalloc((void **)&d1, ROWS*COLS*size);
  hipMalloc((void **)&d2, ROWS*COLS*size);
  hipMalloc((void **)&d3, ROWS*COLS*size);
  hipMemcpy(d1, a, ROWS*COLS*size, hipMemcpyHostToDevice);
  hipMemcpy(d2, b, ROWS*COLS*size, hipMemcpyHostToDevice);

  // add<<<ROWS,COLS>>>(d1, d2, d3); //use with F1
  dim3 blocks_per_grid(COLS, ROWS); //2-D grid
  add<<<blocks_per_grid,1>>>(d1, d2, d3); //use with F2

  hipMemcpy(c, d3, ROWS*COLS*size, hipMemcpyDeviceToHost);
  hipFree(d1); hipFree(d2); hipFree(d3);
  for (int i=0; i<ROWS; i++){
    for (int j=0; j<COLS; j++){
      printf("Sum of %d and %d = %d\n", a[i][j], b[i][j], c[i][j]);
    }
  }
  return 0;
}
